#include "hip/hip_runtime.h"
#include "MainWindow.cuh"
#include <iostream>
#include <string>

#define WIN32_LEAN_AND_MEAN

#include <Windows.h>

#include <hip/hip_runtime.h>
#include <>

#define CHECK_CUDA_ERROR(error, msg) {											\
	if(error != hipSuccess) {													\
		std::cerr << msg << std::endl << hipGetErrorString(error) << std::endl;\
		return false;															\
	}																			\
}

__global__ void PopulateArrayWithIndexed(uint32_t* out, int n)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;

	if(index < n)
		out[index] = ((uint32_t)(index % WIDTH) << 16 | (uint32_t)(index / WIDTH));
}

__global__ void Mandelbrot(uint32_t* in, uint32_t* out, 
	double centerX, double centerY, 
	double cmplxCenterX, double cmplxCenterY, 
	double pixelSize, 
	int maxIteration, int maxValue,
	int n)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = 1;

	double x, cx;
	double y, cy;
	double tempX;

	if(index < n)
	{
		x = 0;
		y = 0;

		cx = (((in[index] & 0xFFFF0000) >> 16) - centerX) * pixelSize + cmplxCenterX;
		cy = (((in[index] & 0x0000FFFF) >> 0) - centerY) * pixelSize - cmplxCenterY;

		for (uint32_t j = 0; j < maxIteration; j++)
		{
			tempX = x * x - y * y + cx;
			y = 2 * x * y + cy;
			x = tempX;

			if (x * x + y * y >= maxValue)
			{
				out[index] = 0xFFFFFFFF;
				return;
			}
		}

		out[index] = 0x000000FF;
	}
}


MainWindow::MainWindow() :
	sf::IWindow(sf::Vector2u(1000, 800), 
		sf::UnitVector2i * SDL_WINDOWPOS_UNDEFINED,
		"Mandelbrot"),
	pRender(nullptr)
{

}

void MainWindow::SetFunction(FractalSequence func)
{
	pFunction = func;
}

bool MainWindow::OnCreate()
{
	pRender = SDL_CreateTexture(m_pRenderer, 
		SDL_PIXELFORMAT_RGBA8888, SDL_TEXTUREACCESS_STREAMING, 
		WIDTH, HEIGHT
	);
	if (pRender == nullptr)
	{
		std::cerr << "Failed to initialize Render Texture: " << SDL_GetError() << std::endl;
		return false;
	}

	SDL_SetRenderDrawColor(m_pRenderer, 0, 0, 0, 255);

	return true;
}

bool MainWindow::GetMandelbrotColors(Uint32** pixels)
{
	constexpr int ARR_SIZE = WIDTH * HEIGHT;

	// Calculate / set thread/block size
	constexpr int threadsPerBlock = 256;
	constexpr int blocksPerGrid = (ARR_SIZE + threadsPerBlock - 1) / threadsPerBlock;

	hipError_t err;

	// Create device memory for screen indices
	if (cuda_screen == NULL)
	{
		err = hipMalloc((void**)&cuda_screen, ARR_SIZE * sizeof(uint32_t));
		CHECK_CUDA_ERROR(err, "Failed to create array on device");

		// Call device kernel to fill array
		PopulateArrayWithIndexed<<<blocksPerGrid, threadsPerBlock>>>(cuda_screen, ARR_SIZE);
		err = hipGetLastError();
		CHECK_CUDA_ERROR(err, "Failed to launch kernel: ");
	}

	// Create device memory for color data;
	if (cuda_colors == NULL)
	{
		err = hipMalloc((void**)&cuda_colors, ARR_SIZE * sizeof(uint32_t));
		CHECK_CUDA_ERROR(err, "Failed to create array on device");
	}

	// Call device kernel to calculate mandelbrot colors for each pixel
	Mandelbrot<<<blocksPerGrid, threadsPerBlock>>>(cuda_screen, cuda_colors,
		WIDTH / 2, HEIGHT / 2,
		-0.77568377f, 0.13646737,
		xInterval / WIDTH,
		1000, 100000,
		ARR_SIZE);
	err = hipGetLastError();
	CHECK_CUDA_ERROR(err, "Failed to launch kernel: ");

	// Free given memory to avoid memory leaks
	if (*pixels == nullptr)
	{
		*pixels = (Uint32*)malloc(ARR_SIZE * sizeof(Uint32));
	}

	err = hipMemcpy(*pixels, cuda_colors, ARR_SIZE * sizeof(Uint32), hipMemcpyDeviceToHost);
	CHECK_CUDA_ERROR(err, "Failed to memcpy from device to host");

	return true;
}

bool MainWindow::OnUpdate(double frametime)
{
	SDL_SetWindowTitle(m_pWindow, (std::to_string(1.f / frametime) + std::string(" FPS")).c_str());

	static int pitch = 0;
	SDL_LockTexture(pRender, NULL, (void**)&pPixels, &pitch);

	if (!GetMandelbrotColors(&pPixels))
	{
		SDL_UnlockTexture(pRender);
		return false;
	}

	SDL_UnlockTexture(pRender);

	xInterval -= frametime * xInterval * 0.5;

	return true;
}

void MainWindow::OnRender(SDL_Renderer* renderer)
{
	SDL_RenderClear(m_pRenderer);
	SDL_RenderCopy(m_pRenderer, pRender, NULL, NULL);
}

void MainWindow::OnClose()
{
	//SDL_DestroyTexture(pRender);
	
	//free(pPixels);
	
	//hipError_t err;

	// Free device memory for color data
	// err = hipFree(cuda_colors);

	// Free device memory for screen indices
	// err = hipFree((void*)cuda_screen);
}

fComplex64 MainWindow::MapComplex(const fComplex64& value, const SDL_Rect& from, const SDL_Rect& to)
{
	fComplex64 ret(
		(value.real() - from.x) * (to.w - to.x) / (from.w - from.x) + to.x,
		(value.imag() - from.y) * (to.h - to.y) / (from.h - from.y) + to.y
	);
	return ret;
}

fComplex64 MainWindow::MapComplex(const fComplex64& value, const fComplex64& centerPoint, double pixelSize)
{
	return ((value - centerPoint) * pixelSize);
}
